#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <petscdevice_cuda.h>
#include <../src/mat/impls/sell/seq/sell.h> /*I   "petscmat.h"  I*/

#define SLICE_HEIGHT 16

typedef struct {
  PetscInt  *colidx; /* column index */
  MatScalar *val;
  PetscInt  *sliidx;
  PetscInt   nonzerostate;
  PetscInt   kernelchoice;
  PetscInt   blocky;
} Mat_SeqSELLCUDA;

static PetscErrorCode MatSeqSELLCUDA_Destroy(Mat_SeqSELLCUDA **cudastruct)
{
  PetscFunctionBegin;
  if (*cudastruct) {
    if ((*cudastruct)->colidx) { PetscCallCUDA(hipFree((*cudastruct)->colidx)); }
    if ((*cudastruct)->val) { PetscCallCUDA(hipFree((*cudastruct)->val)); }
    if ((*cudastruct)->sliidx) { PetscCallCUDA(hipFree((*cudastruct)->sliidx)); }
    PetscCall(PetscFree(*cudastruct));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatSeqSELLCUDACopyToGPU(Mat A)
{
  Mat_SeqSELLCUDA *cudastruct = (Mat_SeqSELLCUDA *)A->spptr;
  Mat_SeqSELL     *a          = (Mat_SeqSELL *)A->data;

  PetscFunctionBegin;
  if (A->offloadmask == PETSC_OFFLOAD_UNALLOCATED || A->offloadmask == PETSC_OFFLOAD_CPU) {
    PetscCall(PetscLogEventBegin(MAT_CUDACopyToGPU, A, 0, 0, 0));
    if (A->assembled && A->nonzerostate == cudastruct->nonzerostate) {
      /* copy values only */
      PetscCallCUDA(hipMemcpy(cudastruct->val, a->val, a->sliidx[a->totalslices] * sizeof(MatScalar), hipMemcpyHostToDevice));
      PetscCall(PetscLogCpuToGpu(a->sliidx[a->totalslices] * (sizeof(MatScalar))));
    } else {
      if (cudastruct->colidx) { PetscCallCUDA(hipFree(cudastruct->colidx)); }
      if (cudastruct->val) { PetscCallCUDA(hipFree(cudastruct->val)); }
      if (cudastruct->sliidx) { PetscCallCUDA(hipFree(cudastruct->sliidx)); }
      PetscCallCUDA(hipMalloc((void **)&(cudastruct->colidx), a->maxallocmat * sizeof(PetscInt)));
      PetscCallCUDA(hipMalloc((void **)&(cudastruct->val), a->maxallocmat * sizeof(MatScalar)));
      /* copy values, nz or maxallocmat? */
      PetscCallCUDA(hipMemcpy(cudastruct->colidx, a->colidx, a->sliidx[a->totalslices] * sizeof(PetscInt), hipMemcpyHostToDevice));
      PetscCallCUDA(hipMemcpy(cudastruct->val, a->val, a->sliidx[a->totalslices] * sizeof(MatScalar), hipMemcpyHostToDevice));

      PetscCallCUDA(hipMalloc((void **)&(cudastruct->sliidx), (a->totalslices + 1) * sizeof(PetscInt)));
      PetscCallCUDA(hipMemcpy(cudastruct->sliidx, a->sliidx, (a->totalslices + 1) * sizeof(PetscInt), hipMemcpyHostToDevice));
      PetscCall(PetscLogCpuToGpu(a->sliidx[a->totalslices] * (sizeof(MatScalar) + sizeof(PetscInt)) + (a->totalslices + 1) * sizeof(PetscInt)));
      cudastruct->nonzerostate = A->nonzerostate;
    }
    PetscCallCUDA(WaitForCUDA());
    PetscCall(PetscLogEventEnd(MAT_CUDACopyToGPU, A, 0, 0, 0));
    A->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

__global__ void matmult_seqsell_basic_kernel(PetscInt nrows, PetscInt sliceheight, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, PetscScalar *y)
{
  PetscInt  i, row, slice_id, row_in_slice;
  MatScalar sum;
  /* one thread per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / sliceheight;
    row_in_slice = row % sliceheight;
    sum          = 0.0;
    for (i = sliidx[slice_id] + row_in_slice; i < sliidx[slice_id + 1]; i += sliceheight) sum += aval[i] * x[acolidx[i]];
    y[row] = sum;
  }
}

__global__ void matmultadd_seqsell_basic_kernel(PetscInt nrows, PetscInt sliceheight, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, const PetscScalar *y, PetscScalar *z)
{
  PetscInt  i, row, slice_id, row_in_slice;
  MatScalar sum;
  /* one thread per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / sliceheight;
    row_in_slice = row % sliceheight;
    sum          = 0.0;
    for (i = sliidx[slice_id] + row_in_slice; i < sliidx[slice_id + 1]; i += sliceheight) sum += aval[i] * x[acolidx[i]];
    z[row] = y[row] + sum;
  }
}

/* use 1 block per slice, suitable for large slice width */
template <int BLOCKY>
__global__ void matmult_seqsell_tiled_kernel9(PetscInt nrows, PetscInt sliceheight, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, PetscScalar *y)
{
  __shared__ MatScalar shared[32][BLOCKY];
  PetscInt             i, row, slice_id = blockIdx.x;
  int                  tid = threadIdx.x + threadIdx.y * 32;
  /* transposed index */
  int         tidx = tid % BLOCKY;
  int         tidy = tid / BLOCKY;
  PetscScalar t    = 0.0;

  row = slice_id * sliceheight + threadIdx.x % sliceheight;
  if (row < nrows) {
    for (i = sliidx[slice_id] + threadIdx.x + 32 * threadIdx.y; i < sliidx[slice_id + 1]; i += 32 * BLOCKY) t += aval[i] * x[acolidx[i]];
  }
#pragma unroll
  for (int offset = 16; offset >= sliceheight; offset /= 2) { t += __shfl_down_sync(0xffffffff, t, offset); }
  /* transpose layout to reduce each row using warp shfl */
  if (threadIdx.x < sliceheight) shared[threadIdx.x][threadIdx.y] = t;
  __syncthreads();
  if (tidy < sliceheight) t = shared[tidy][tidx];
#pragma unroll
  for (int offset = BLOCKY / 2; offset > 0; offset /= 2) { t += __shfl_down_sync(0xffffffff, t, offset, BLOCKY); }
  if (tidx == 0 && tidy < sliceheight) { shared[0][tidy] = t; }
  __syncthreads();
  if (row < nrows && threadIdx.y == 0 && threadIdx.x < sliceheight) { y[row] = shared[0][threadIdx.x]; }
}

/* use 1 block per slice, suitable for large slice width */
template <int BLOCKY>
__global__ void matmultadd_seqsell_tiled_kernel9(PetscInt nrows, PetscInt sliceheight, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, const PetscScalar *y, PetscScalar *z)
{
  __shared__ MatScalar shared[32][BLOCKY];
  PetscInt             i, row, slice_id = blockIdx.x;
  int                  tid = threadIdx.x + threadIdx.y * 32;
  /* transposed index */
  int         tidx = tid % BLOCKY;
  int         tidy = tid / BLOCKY;
  PetscScalar t    = 0.0;

  row = slice_id * sliceheight + threadIdx.x % sliceheight;
  if (row < nrows) {
    for (i = sliidx[slice_id] + threadIdx.x + 32 * threadIdx.y; i < sliidx[slice_id + 1]; i += 32 * BLOCKY) t += aval[i] * x[acolidx[i]];
  }
#pragma unroll
  for (int offset = 16; offset >= sliceheight; offset /= 2) { t += __shfl_down_sync(0xffffffff, t, offset); }
  /* transpose layout to reduce each row using warp shfl */
  if (threadIdx.x < sliceheight) shared[threadIdx.x][threadIdx.y] = t;
  __syncthreads();
  if (tidy < sliceheight) t = shared[tidy][tidx];
#pragma unroll
  for (int offset = BLOCKY / 2; offset > 0; offset /= 2) { t += __shfl_down_sync(0xffffffff, t, offset, BLOCKY); }
  if (tidx == 0 && tidy < sliceheight) { shared[0][tidy] = t; }
  __syncthreads();
  if (row < nrows && threadIdx.y == 0 && threadIdx.x < sliceheight) { z[row] = y[row] + shared[0][threadIdx.x]; }
}

/* use 1 warp per slice, suitable for small slice width */
__global__ void matmult_seqsell_tiled_kernel7(PetscInt nrows, PetscInt sliceheight, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, PetscScalar *y)
{
  PetscInt i, row, slice_id;
  slice_id = blockIdx.x * blockDim.y + threadIdx.y;
  row      = slice_id * sliceheight + threadIdx.x % sliceheight;
  double t = 0.0;
  if (row < nrows) {
    for (i = sliidx[slice_id] + threadIdx.x; i < sliidx[slice_id + 1]; i += 32) t += aval[i] * x[acolidx[i]];
  }
#pragma unroll
  for (int offset = 16; offset >= sliceheight; offset /= 2) { t += __shfl_down_sync(0xffffffff, t, offset); }
  if (row < nrows && threadIdx.x < sliceheight) { y[row] = t; }
}

/* use 1 warp per slice, suitable for small slice width */
__global__ void matmultadd_seqsell_tiled_kernel7(PetscInt nrows, PetscInt sliceheight, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, const PetscScalar *y, PetscScalar *z)
{
  PetscInt i, row, slice_id;
  slice_id = blockIdx.x * blockDim.y + threadIdx.y;
  row      = slice_id * sliceheight + threadIdx.x % sliceheight;
  double t = 0.0;
  if (row < nrows) {
    for (i = sliidx[slice_id] + threadIdx.x; i < sliidx[slice_id + 1]; i += 32) t += aval[i] * x[acolidx[i]];
  }
#pragma unroll
  for (int offset = 16; offset >= sliceheight; offset /= 2) { t += __shfl_down_sync(0xffffffff, t, offset); }
  if (row < nrows && threadIdx.x < sliceheight) { z[row] = y[row] + t; }
}

/***********  Kernel 2-6  are tied to slice height 16. They are kept only for performance comparison  **********/

__global__ void matmult_seqsell_tiled_kernel6(PetscInt nrows, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, PetscScalar *y)
{
  __shared__ MatScalar shared[512];
  PetscInt             i, row, slice_id, row_in_slice;
  /* multiple threads per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / SLICE_HEIGHT;
    row_in_slice = row % SLICE_HEIGHT;

    shared[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
    for (i = sliidx[slice_id] + row_in_slice + SLICE_HEIGHT * threadIdx.y; i < sliidx[slice_id + 1]; i += SLICE_HEIGHT * blockDim.y) shared[threadIdx.y * blockDim.x + threadIdx.x] += aval[i] * x[acolidx[i]];
    __syncthreads();
    if (threadIdx.y < 16) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 16) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 8) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 8) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 4) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 4) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 2) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 2) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 1) {
      shared[threadIdx.x] += shared[blockDim.x + threadIdx.x];
      y[row] = shared[threadIdx.x];
    }
  }
}

__global__ void matmult_seqsell_tiled_kernel5(PetscInt nrows, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, PetscScalar *y)
{
  __shared__ MatScalar shared[512];
  PetscInt             i, row, slice_id, row_in_slice;
  /* multiple threads per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / SLICE_HEIGHT;
    row_in_slice = row % SLICE_HEIGHT;

    shared[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
    for (i = sliidx[slice_id] + row_in_slice + SLICE_HEIGHT * threadIdx.y; i < sliidx[slice_id + 1]; i += SLICE_HEIGHT * blockDim.y) shared[threadIdx.y * blockDim.x + threadIdx.x] += aval[i] * x[acolidx[i]];
    __syncthreads();
    if (threadIdx.y < 8) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 8) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 4) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 4) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 2) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 2) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 1) {
      shared[threadIdx.x] += shared[blockDim.x + threadIdx.x];
      y[row] = shared[threadIdx.x];
    }
  }
}

__global__ void matmult_seqsell_tiled_kernel4(PetscInt nrows, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, PetscScalar *y)
{
  __shared__ MatScalar shared[512];
  PetscInt             i, row, slice_id, row_in_slice;
  /* multiple threads per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / SLICE_HEIGHT;
    row_in_slice = row % SLICE_HEIGHT;

    shared[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
    for (i = sliidx[slice_id] + row_in_slice + SLICE_HEIGHT * threadIdx.y; i < sliidx[slice_id + 1]; i += SLICE_HEIGHT * blockDim.y) shared[threadIdx.y * blockDim.x + threadIdx.x] += aval[i] * x[acolidx[i]];
    __syncthreads();
    if (threadIdx.y < 4) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 4) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 2) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 2) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 1) {
      shared[threadIdx.x] += shared[blockDim.x + threadIdx.x];
      y[row] = shared[threadIdx.x];
    }
  }
}

__global__ void matmult_seqsell_tiled_kernel3(PetscInt nrows, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, PetscScalar *y)
{
  __shared__ MatScalar shared[512];
  PetscInt             i, row, slice_id, row_in_slice;
  /* multiple threads per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / SLICE_HEIGHT;
    row_in_slice = row % SLICE_HEIGHT;

    shared[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
    for (i = sliidx[slice_id] + row_in_slice + SLICE_HEIGHT * threadIdx.y; i < sliidx[slice_id + 1]; i += SLICE_HEIGHT * blockDim.y) shared[threadIdx.y * blockDim.x + threadIdx.x] += aval[i] * x[acolidx[i]];
    __syncthreads();
    if (threadIdx.y < 2) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 2) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 1) {
      shared[threadIdx.x] += shared[blockDim.x + threadIdx.x];
      y[row] = shared[threadIdx.x];
    }
  }
}

__global__ void matmult_seqsell_tiled_kernel2(PetscInt nrows, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, PetscScalar *y)
{
  __shared__ MatScalar shared[512];
  PetscInt             i, row, slice_id, row_in_slice;
  /* multiple threads per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / SLICE_HEIGHT;
    row_in_slice = row % SLICE_HEIGHT;

    shared[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
    for (i = sliidx[slice_id] + row_in_slice + SLICE_HEIGHT * threadIdx.y; i < sliidx[slice_id + 1]; i += SLICE_HEIGHT * blockDim.y) shared[threadIdx.y * blockDim.x + threadIdx.x] += aval[i] * x[acolidx[i]];
    __syncthreads();
    if (threadIdx.y < 1) {
      shared[threadIdx.x] += shared[blockDim.x + threadIdx.x];
      y[row] = shared[threadIdx.x];
    }
  }
}

__global__ void matmultadd_seqsell_tiled_kernel6(PetscInt nrows, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, const PetscScalar *y, PetscScalar *z)
{
  __shared__ MatScalar shared[512];
  PetscInt             i, row, slice_id, row_in_slice;
  /* multiple threads per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / SLICE_HEIGHT;
    row_in_slice = row % SLICE_HEIGHT;

    shared[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
    for (i = sliidx[slice_id] + row_in_slice + SLICE_HEIGHT * threadIdx.y; i < sliidx[slice_id + 1]; i += SLICE_HEIGHT * blockDim.y) shared[threadIdx.y * blockDim.x + threadIdx.x] += aval[i] * x[acolidx[i]];
    __syncthreads();
    if (threadIdx.y < 16) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 16) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 8) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 8) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 4) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 4) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 2) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 2) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 1) {
      shared[threadIdx.x] += shared[blockDim.x + threadIdx.x];
      z[row] = y[row] + shared[threadIdx.x];
    }
  }
}

__global__ void matmultadd_seqsell_tiled_kernel5(PetscInt nrows, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, const PetscScalar *y, PetscScalar *z)
{
  __shared__ MatScalar shared[512];
  PetscInt             i, row, slice_id, row_in_slice;
  /* multiple threads per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / SLICE_HEIGHT;
    row_in_slice = row % SLICE_HEIGHT;

    shared[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
    for (i = sliidx[slice_id] + row_in_slice + SLICE_HEIGHT * threadIdx.y; i < sliidx[slice_id + 1]; i += SLICE_HEIGHT * blockDim.y) shared[threadIdx.y * blockDim.x + threadIdx.x] += aval[i] * x[acolidx[i]];
    __syncthreads();
    if (threadIdx.y < 8) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 8) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 4) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 4) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 2) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 2) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 1) {
      shared[threadIdx.x] += shared[blockDim.x + threadIdx.x];
      z[row] = y[row] + shared[threadIdx.x];
    }
  }
}

__global__ void matmultadd_seqsell_tiled_kernel4(PetscInt nrows, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, const PetscScalar *y, PetscScalar *z)
{
  __shared__ MatScalar shared[512];
  PetscInt             i, row, slice_id, row_in_slice;
  /* multiple threads per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / SLICE_HEIGHT;
    row_in_slice = row % SLICE_HEIGHT;

    shared[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
    for (i = sliidx[slice_id] + row_in_slice + SLICE_HEIGHT * threadIdx.y; i < sliidx[slice_id + 1]; i += SLICE_HEIGHT * blockDim.y) shared[threadIdx.y * blockDim.x + threadIdx.x] += aval[i] * x[acolidx[i]];
    __syncthreads();
    if (threadIdx.y < 4) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 4) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 2) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 2) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 1) {
      shared[threadIdx.x] += shared[blockDim.x + threadIdx.x];
      z[row] = y[row] + shared[threadIdx.x];
    }
  }
}

__global__ void matmultadd_seqsell_tiled_kernel3(PetscInt nrows, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, const PetscScalar *y, PetscScalar *z)
{
  __shared__ MatScalar shared[512];
  PetscInt             i, row, slice_id, row_in_slice;
  /* multiple threads per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / SLICE_HEIGHT;
    row_in_slice = row % SLICE_HEIGHT;

    shared[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
    for (i = sliidx[slice_id] + row_in_slice + SLICE_HEIGHT * threadIdx.y; i < sliidx[slice_id + 1]; i += SLICE_HEIGHT * blockDim.y) shared[threadIdx.y * blockDim.x + threadIdx.x] += aval[i] * x[acolidx[i]];
    __syncthreads();
    if (threadIdx.y < 2) { shared[threadIdx.y * blockDim.x + threadIdx.x] += shared[(threadIdx.y + 2) * blockDim.x + threadIdx.x]; }
    __syncthreads();
    if (threadIdx.y < 1) {
      shared[threadIdx.x] += shared[blockDim.x + threadIdx.x];
      z[row] = y[row] + shared[threadIdx.x];
    }
  }
}

__global__ void matmultadd_seqsell_tiled_kernel2(PetscInt nrows, const PetscInt *acolidx, const MatScalar *aval, const PetscInt *sliidx, const PetscScalar *x, const PetscScalar *y, PetscScalar *z)
{
  __shared__ MatScalar shared[512];
  PetscInt             i, row, slice_id, row_in_slice;
  /* multiple threads per row. */
  row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < nrows) {
    slice_id     = row / SLICE_HEIGHT;
    row_in_slice = row % SLICE_HEIGHT;

    shared[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
    for (i = sliidx[slice_id] + row_in_slice + SLICE_HEIGHT * threadIdx.y; i < sliidx[slice_id + 1]; i += SLICE_HEIGHT * blockDim.y) shared[threadIdx.y * blockDim.x + threadIdx.x] += aval[i] * x[acolidx[i]];
    __syncthreads();
    if (threadIdx.y < 1) {
      shared[threadIdx.x] += shared[blockDim.x + threadIdx.x];
      z[row] = y[row] + shared[threadIdx.x];
    }
  }
}

PetscErrorCode MatMult_SeqSELLCUDA(Mat A, Vec xx, Vec yy)
{
  Mat_SeqSELL       *a          = (Mat_SeqSELL *)A->data;
  Mat_SeqSELLCUDA   *cudastruct = (Mat_SeqSELLCUDA *)A->spptr;
  PetscScalar       *y;
  const PetscScalar *x;
  PetscInt           nrows = A->rmap->n, sliceheight = a->sliceheight;
  MatScalar         *aval;
  PetscInt          *acolidx;
  PetscInt          *sliidx;
  PetscInt           nblocks, blocksize = 512; /* blocksize must be multiple of SLICE_HEIGHT*32 */
  dim3               block2(256, 2), block4(128, 4), block8(64, 8), block16(32, 16), block32(16, 32);

  PetscFunctionBegin;
  PetscCheck(32 % sliceheight == 0, PETSC_COMM_SELF, PETSC_ERR_SUP, "The kernel requires a slice height be a divisor of 32, but the input matrix has a slice height of %" PetscInt_FMT, sliceheight);
  PetscCall(MatSeqSELLCUDACopyToGPU(A));
  /* cudastruct may not be available until MatSeqSELLCUDACopyToGPU() is called */
  aval    = cudastruct->val;
  acolidx = cudastruct->colidx;
  sliidx  = cudastruct->sliidx;

  PetscCall(VecCUDAGetArrayRead(xx, &x));
  PetscCall(VecCUDAGetArrayWrite(yy, &y));
  PetscCall(PetscLogGpuTimeBegin());

  switch (cudastruct->kernelchoice) {
  case 9:
    nblocks = 1 + (nrows - 1) / sliceheight;
    if (cudastruct->blocky == 2) {
      matmult_seqsell_tiled_kernel9<2><<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    } else if (cudastruct->blocky == 4) {
      matmult_seqsell_tiled_kernel9<4><<<nblocks, dim3(32, 4)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    } else if (cudastruct->blocky == 8) {
      matmult_seqsell_tiled_kernel9<8><<<nblocks, dim3(32, 8)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    } else if (cudastruct->blocky == 16) {
      matmult_seqsell_tiled_kernel9<16><<<nblocks, dim3(32, 16)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    } else if (cudastruct->blocky == 32) {
      matmult_seqsell_tiled_kernel9<32><<<nblocks, dim3(32, 32)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    } else {
      matmult_seqsell_tiled_kernel9<2><<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    }
    break;
  case 7:
    nblocks = 1 + (nrows - 1) / (2 * sliceheight);
    if (cudastruct->blocky == 2) {
      matmult_seqsell_tiled_kernel7<<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    } else if (cudastruct->blocky == 4) {
      matmult_seqsell_tiled_kernel7<<<nblocks, dim3(32, 4)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    } else if (cudastruct->blocky == 8) {
      matmult_seqsell_tiled_kernel7<<<nblocks, dim3(32, 8)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    } else if (cudastruct->blocky == 16) {
      matmult_seqsell_tiled_kernel7<<<nblocks, dim3(32, 16)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    } else if (cudastruct->blocky == 32) {
      matmult_seqsell_tiled_kernel7<<<nblocks, dim3(32, 32)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    } else {
      matmult_seqsell_tiled_kernel7<<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    }
    break;
  case 6:
    nblocks = 1 + (nrows - 1) / (blocksize / 32); /* 1 slice per block if blocksize=512 */
    matmult_seqsell_tiled_kernel6<<<nblocks, block32>>>(nrows, acolidx, aval, sliidx, x, y);
    break;
  case 5:
    nblocks = 1 + (nrows - 1) / (blocksize / 16); /* 2 slices per block if blocksize=512*/
    matmult_seqsell_tiled_kernel5<<<nblocks, block16>>>(nrows, acolidx, aval, sliidx, x, y);
    break;
  case 4:
    nblocks = 1 + (nrows - 1) / (blocksize / 8); /* 4 slices per block if blocksize=512 */
    matmult_seqsell_tiled_kernel4<<<nblocks, block8>>>(nrows, acolidx, aval, sliidx, x, y);
    break;
  case 3:
    nblocks = 1 + (nrows - 1) / (blocksize / 4); /* 8 slices per block if blocksize=512 */
    matmult_seqsell_tiled_kernel3<<<nblocks, block4>>>(nrows, acolidx, aval, sliidx, x, y);
    break;
  case 2: /* 16 slices per block if blocksize=512 */
    nblocks = 1 + (nrows - 1) / (blocksize / 2);
    matmult_seqsell_tiled_kernel2<<<nblocks, block2>>>(nrows, acolidx, aval, sliidx, x, y);
    break;
  case 1: /* 32 slices per block if blocksize=512 */
    nblocks = 1 + (nrows - 1) / blocksize;
    matmult_seqsell_basic_kernel<<<nblocks, blocksize>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    break;
  case 0:
    if (sliceheight * a->maxslicewidth > 20800) { /* important threshold */
      nblocks = 1 + (nrows - 1) / sliceheight;
      matmult_seqsell_tiled_kernel9<32><<<nblocks, dim3(32, 32)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
    } else {
      PetscInt avgslicesize = sliceheight * a->avgslicewidth;
      if (avgslicesize <= 96) {
        nblocks = 1 + (nrows - 1) / (2 * sliceheight); /* two slices per block */
        matmult_seqsell_tiled_kernel7<<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
      } else if (avgslicesize <= 432) {
        nblocks = 1 + (nrows - 1) / sliceheight;
        matmult_seqsell_tiled_kernel9<2><<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
      } else if (avgslicesize <= 2400) {
        nblocks = 1 + (nrows - 1) / sliceheight;
        matmult_seqsell_tiled_kernel9<8><<<nblocks, dim3(32, 8)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
      } else {
        nblocks = 1 + (nrows - 1) / sliceheight;
        matmult_seqsell_tiled_kernel9<16><<<nblocks, dim3(32, 16)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y);
      }
    }
    break;
  }
  PetscCall(PetscLogGpuTimeEnd());
  PetscCall(VecCUDARestoreArrayRead(xx, &x));
  PetscCall(VecCUDARestoreArrayWrite(yy, &y));
  PetscCall(PetscLogGpuFlops(2.0 * a->nz - a->nonzerorowcnt));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode MatMultAdd_SeqSELLCUDA(Mat A, Vec xx, Vec yy, Vec zz)
{
  Mat_SeqSELL       *a          = (Mat_SeqSELL *)A->data;
  Mat_SeqSELLCUDA   *cudastruct = (Mat_SeqSELLCUDA *)A->spptr;
  PetscScalar       *z;
  const PetscScalar *y, *x;
  PetscInt           nrows = A->rmap->n, sliceheight = a->sliceheight;
  MatScalar         *aval    = cudastruct->val;
  PetscInt          *acolidx = cudastruct->colidx;
  PetscInt          *sliidx  = cudastruct->sliidx;

  PetscFunctionBegin;
  PetscCheck(sliceheight == 16, PETSC_COMM_SELF, PETSC_ERR_SUP, "The kernel requires a slice height of 16, but the input matrix has a slice height of %" PetscInt_FMT, sliceheight);
  PetscCall(MatSeqSELLCUDACopyToGPU(A));
  if (a->nz) {
    PetscInt nblocks, blocksize = 512;
    dim3     block2(256, 2), block4(128, 4), block8(64, 8), block16(32, 16), block32(16, 32);
    PetscCall(VecCUDAGetArrayRead(xx, &x));
    PetscCall(VecCUDAGetArrayRead(yy, &y));
    PetscCall(VecCUDAGetArrayWrite(zz, &z));
    PetscCall(PetscLogGpuTimeBegin());

    switch (cudastruct->kernelchoice) {
    case 9:
      nblocks = 1 + (nrows - 1) / sliceheight;
      if (cudastruct->blocky == 2) {
        matmultadd_seqsell_tiled_kernel9<2><<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      } else if (cudastruct->blocky == 4) {
        matmultadd_seqsell_tiled_kernel9<4><<<nblocks, dim3(32, 4)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      } else if (cudastruct->blocky == 8) {
        matmultadd_seqsell_tiled_kernel9<8><<<nblocks, dim3(32, 8)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      } else if (cudastruct->blocky == 16) {
        matmultadd_seqsell_tiled_kernel9<16><<<nblocks, dim3(32, 16)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      } else if (cudastruct->blocky == 32) {
        matmultadd_seqsell_tiled_kernel9<32><<<nblocks, dim3(32, 32)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      } else {
        matmultadd_seqsell_tiled_kernel9<2><<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      }
      break;
    case 7:
      nblocks = 1 + (nrows - 1) / (2 * sliceheight);
      if (cudastruct->blocky == 2) {
        matmultadd_seqsell_tiled_kernel7<<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      } else if (cudastruct->blocky == 4) {
        matmultadd_seqsell_tiled_kernel7<<<nblocks, dim3(32, 4)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      } else if (cudastruct->blocky == 8) {
        matmultadd_seqsell_tiled_kernel7<<<nblocks, dim3(32, 8)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      } else if (cudastruct->blocky == 16) {
        matmultadd_seqsell_tiled_kernel7<<<nblocks, dim3(32, 16)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      } else if (cudastruct->blocky == 32) {
        matmultadd_seqsell_tiled_kernel7<<<nblocks, dim3(32, 32)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      } else {
        matmultadd_seqsell_tiled_kernel7<<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      }
      break;
    case 6:
      nblocks = 1 + (nrows - 1) / (blocksize / 32);
      matmultadd_seqsell_tiled_kernel6<<<nblocks, block32>>>(nrows, acolidx, aval, sliidx, x, y, z);
      break;
    case 5:
      nblocks = 1 + (nrows - 1) / (blocksize / 16);
      matmultadd_seqsell_tiled_kernel5<<<nblocks, block16>>>(nrows, acolidx, aval, sliidx, x, y, z);
      break;
    case 4:
      nblocks = 1 + (nrows - 1) / (blocksize / 8);
      matmultadd_seqsell_tiled_kernel4<<<nblocks, block8>>>(nrows, acolidx, aval, sliidx, x, y, z);
      break;
    case 3:
      nblocks = 1 + (nrows - 1) / (blocksize / 4);
      matmultadd_seqsell_tiled_kernel3<<<nblocks, block4>>>(nrows, acolidx, aval, sliidx, x, y, z);
      break;
    case 2:
      nblocks = 1 + (nrows - 1) / (blocksize / 2);
      matmultadd_seqsell_tiled_kernel2<<<nblocks, block2>>>(nrows, acolidx, aval, sliidx, x, y, z);
      break;
    case 1:
      nblocks = 1 + (nrows - 1) / blocksize;
      matmultadd_seqsell_basic_kernel<<<nblocks, blocksize>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      break;
    case 0:
      if (sliceheight * a->maxslicewidth > 20800) {
        nblocks = 1 + (nrows - 1) / sliceheight;
        matmultadd_seqsell_tiled_kernel9<32><<<nblocks, dim3(32, 32)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
      } else {
        PetscInt avgslicesize = sliceheight * a->avgslicewidth;
        if (avgslicesize <= 96) {
          nblocks = 1 + (nrows - 1) / (2 * sliceheight); /* two slices per block */
          matmultadd_seqsell_tiled_kernel7<<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
        } else if (avgslicesize <= 432) {
          nblocks = 1 + (nrows - 1) / sliceheight;
          matmultadd_seqsell_tiled_kernel9<2><<<nblocks, dim3(32, 2)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
        } else if (avgslicesize <= 2400) {
          nblocks = 1 + (nrows - 1) / sliceheight;
          matmultadd_seqsell_tiled_kernel9<8><<<nblocks, dim3(32, 8)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
        } else {
          nblocks = 1 + (nrows - 1) / sliceheight;
          matmultadd_seqsell_tiled_kernel9<16><<<nblocks, dim3(32, 16)>>>(nrows, sliceheight, acolidx, aval, sliidx, x, y, z);
        }
      }
      break;
    }
    PetscCall(PetscLogGpuTimeEnd());
    PetscCall(VecCUDARestoreArrayRead(xx, &x));
    PetscCall(VecCUDARestoreArrayRead(yy, &y));
    PetscCall(VecCUDARestoreArrayWrite(zz, &z));
    PetscCall(PetscLogGpuFlops(2.0 * a->nz));
  } else {
    PetscCall(VecCopy(yy, zz));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatSetFromOptions_SeqSELLCUDA(Mat A, PetscOptionItems *PetscOptionsObject)
{
  Mat_SeqSELLCUDA *cudastruct = (Mat_SeqSELLCUDA *)A->spptr;
  PetscInt         kernel, blocky;
  PetscBool        flg;

  PetscFunctionBegin;
  PetscOptionsHeadBegin(PetscOptionsObject, "SeqSELLCUDA options");
  PetscCall(PetscOptionsGetInt(NULL, NULL, "-mat_sell_spmv_cuda_kernel", &kernel, &flg));
  if (flg) {
    PetscCheck(kernel >= 0 && kernel <= 9, PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "Wrong kernel choice: %" PetscInt_FMT " it should be in [0,9]", kernel);
    cudastruct->kernelchoice = kernel;
  }
  PetscCall(PetscOptionsGetInt(NULL, NULL, "-mat_sell_spmv_cuda_blocky", &blocky, &flg));
  if (flg) {
    PetscCheck(blocky == 2 || blocky == 4 || blocky == 8 || blocky == 16 || blocky == 32, PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "Unsupported blocky: %" PetscInt_FMT " it should be in {2,4,8,16,32}", kernel);
    cudastruct->blocky = blocky;
  }
  PetscOptionsHeadEnd();
  PetscFunctionReturn(PETSC_SUCCESS);
}

PETSC_INTERN PetscErrorCode MatAssemblyEnd_SpMV_Preprocessing_Private(Mat A)
{
  Mat_SeqSELL *a = (Mat_SeqSELL *)A->data;

  PetscCall(MatSeqSELLGetAvgSliceWidth(A, &a->avgslicewidth));
  PetscCall(MatSeqSELLGetMaxSliceWidth(A, &a->maxslicewidth));
  PetscCall(MatSeqSELLGetFillRatio(A, &a->fillratio));
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatAssemblyEnd_SeqSELLCUDA(Mat A, MatAssemblyType mode)
{
  PetscFunctionBegin;
  PetscCall(MatAssemblyEnd_SeqSELL(A, mode));
  PetscCall(MatAssemblyEnd_SpMV_Preprocessing_Private(A));
  if (mode == MAT_FLUSH_ASSEMBLY) PetscFunctionReturn(PETSC_SUCCESS);
  if (A->factortype == MAT_FACTOR_NONE) { PetscCall(MatSeqSELLCUDACopyToGPU(A)); }
  A->ops->mult    = MatMult_SeqSELLCUDA;
  A->ops->multadd = MatMultAdd_SeqSELLCUDA;
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatDestroy_SeqSELLCUDA(Mat A)
{
  PetscFunctionBegin;
  if (A->factortype == MAT_FACTOR_NONE) {
    if (A->offloadmask != PETSC_OFFLOAD_UNALLOCATED) { PetscCall(MatSeqSELLCUDA_Destroy((Mat_SeqSELLCUDA **)&A->spptr)); }
  }
  PetscCall(MatDestroy_SeqSELL(A));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PETSC_INTERN PetscErrorCode MatConvert_SeqSELL_SeqSELLCUDA(Mat);
static PetscErrorCode       MatDuplicate_SeqSELLCUDA(Mat A, MatDuplicateOption cpvalues, Mat *B)
{
  PetscFunctionBegin;
  PetscCall(MatDuplicate_SeqSELL(A, cpvalues, B));
  PetscCall(MatConvert_SeqSELL_SeqSELLCUDA(*B));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PETSC_EXTERN PetscErrorCode MatConvert_SeqSELL_SeqSELLCUDA(Mat B)
{
  Mat_SeqSELLCUDA *cudastruct;

  PetscFunctionBegin;
  PetscCall(PetscFree(B->defaultvectype));
  PetscCall(PetscStrallocpy(VECCUDA, &B->defaultvectype));

  if (!B->spptr) {
    if (B->factortype == MAT_FACTOR_NONE) {
      PetscCall(PetscNew(&cudastruct));
      B->spptr = cudastruct;
    }
  }

  B->ops->assemblyend    = MatAssemblyEnd_SeqSELLCUDA;
  B->ops->destroy        = MatDestroy_SeqSELLCUDA;
  B->ops->setfromoptions = MatSetFromOptions_SeqSELLCUDA;
  B->ops->mult           = MatMult_SeqSELLCUDA;
  B->ops->multadd        = MatMultAdd_SeqSELLCUDA;
  B->ops->duplicate      = MatDuplicate_SeqSELLCUDA;

  /* No need to assemble SeqSELL, but need to do the preprocessing for SpMV */
  PetscCall(MatAssemblyEnd_SpMV_Preprocessing_Private(B));

  PetscCall(PetscObjectChangeTypeName((PetscObject)B, MATSEQSELLCUDA));
  B->offloadmask = PETSC_OFFLOAD_UNALLOCATED;
  PetscFunctionReturn(PETSC_SUCCESS);
}

PETSC_EXTERN PetscErrorCode MatCreate_SeqSELLCUDA(Mat B)
{
  PetscFunctionBegin;
  PetscCall(MatCreate_SeqSELL(B));
  PetscCall(MatConvert_SeqSELL_SeqSELLCUDA(B));
  PetscFunctionReturn(PETSC_SUCCESS);
}
